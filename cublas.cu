#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <hipblas.h>

#include "memory.cuh"



int  main (int argc, char** argv) {
  hipblasStatus_t status;
  //cublasInit(); // Removed for transisting to cublas v2 API.

  double *A64 = initializeGroundtruthMat<double>(HA, WA, true, -1);
  double *B64 = initializeGroundtruthMat<double>(HB, WB, true, -1);

  // Initialize matrices A and B (2d arrays) based on the HA/WA and HB/WB to be filled with random data
  float *floatA = initializeFloatFromDouble(HA, WA, A64);
  float *floatB = initializeFloatFromDouble(HB, WB, B64);
  // Create arrays of C64 and it should contain the value of A64*B64, as groundtruth.
  double *C64 = initializeGroundtruthMat<double>(HC, WC, true, -1);

  if( A64 == 0 || B64 == 0 || C64 == 0){
    return EXIT_FAILURE;
  } else {
    // Since Ampere support IEEE-compliant FP64 computations, we use cublas for the computation.
    float *floatC = initializeGroundtruthMat<float>(HC, WC, false, 0);
    float *floatCFromCpu = initializeGroundtruthMat<float>(HC, WC, false, -1);
    float alpha = 1.0f, beta = 0.0f;
    double alpha64 = 1.0, beta64 = 1.0;
    
    hipblasHandle_t handle;
    cublasErrCheck( hipblasCreate(&handle) );
    
    double *devA64, *devB64, *devC64;
    devA64 = initializeDeviceMatFromHostMat(HA, WA, A64);
    devB64 = initializeDeviceMatFromHostMat(HB, WB, B64);
    devC64 = initializeDeviceMatFromHostMat(HC, WC, C64);
    // First compute the ground truth  
    cublasErrCheck(
      hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, HA, WB, WA, &alpha64,
          devA64, HA,
          devB64, HB,
          &beta64,
          devC64, HC)
    );
    // Copy the value back, C64 now stored the groundtruth.
    retrieveDeviceMemory<double>(HC, WC, devC64, C64);
    // Now we can free the devX64 pointers
    freeDevicePointers(devA64, devB64, devC64);


    // Init device memory from host memory
    float *devA, *devB, *devC;
    devA = initializeDeviceMatFromHostMat(HA, WA, floatA);
    devB = initializeDeviceMatFromHostMat(HB, WB, floatB);
    devC = initializeDeviceMatFromHostMat(HC, WC, floatC);

    
    // Matrix to Matrix Multiplication, GEMM
    cublasErrCheck(
      hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, HA, WB, WA, &alpha,
          devA, HA,
          devB, HB,
          &beta,
          devC, HC)
    );

    retrieveDeviceMemory(HC, WC, devC, floatC);

    float* C32;
    C32 = initializeGroundtruthMat<float>(HC, WC, false, 0);
    for (int i = 0; i < HC; i++)
    {
      for (int j = 0; j < WC; j++)
      {
        for (int k = 0; k < WA; k++)
        {
          C32[indexTo1D(i,j,HC)] += A64[indexTo1D(i,k,HA)] * B64[indexTo1D(k,j,HB)];
        }
      }
    }

    // Show the difference.
    int i,j;
    double maxAbsErr = 0;
    double maxRelErr = 0;
    double meanRelErr = 0;
    double meanAbsErr = 0;
    for (i = 0; i < HC; i++)
    {
      for (j = 0; j < WC; j++)
      {
        double diff = std::abs(C32[indexTo1D(i,j,HC)] - floatC[indexTo1D(i,j,HC)]);
        maxAbsErr = std::max(diff, maxAbsErr);
        double relDiff = (C32[indexTo1D(i,j,HC)] == 0 ? 0 : std::abs(diff / C32[indexTo1D(i,j,HC)]));
        maxRelErr = std::max(relDiff, maxRelErr);
        meanAbsErr += diff;
        meanRelErr += relDiff;
      }
    }

    printf("The diffenence of float_using_gpu vs FP32_cpu \n");
    printf("Max Abs diff: %e, Max Rel diff: %e\n", maxAbsErr, maxRelErr);
    printf("Avg Abs diff: %e, Avg Rel diff: %e\n", meanAbsErr/(1.0*HC*WC), meanRelErr/(1.0*HC*WC));


    // printf("==== A ====\n");
    // printMat(floatA, WA, HA);
    // printf("==== B ====\n");
    // printMat(floatB, WB, HB);
    printf("==== C64 ====\n");
    printMat(C64, WC, HC);
    printf("==== C32 ====\n");
    printMat(C32, WC, HC);
    printf("==== C ====\n");
    printMat(floatC, WC, HC);

    freeHostPointers(A64, B64, C64, floatA, floatB, floatC, C32);
    freeDevicePointers(devA, devB, devC);
    
    /* Shutdown */
    cublasErrCheck( hipblasDestroy(handle) );

    return EXIT_SUCCESS;
  }

}

