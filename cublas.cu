#include <stdlib.h>
#include <stdio.h>
#include <hipblas.h>

#include "memory.cuh"

// __host__ void printMatrices(float *A, float *B, float *C){
//   printf("\nMatrix A:\n");
//   printMat(A,WA,HA);
//   printf("\n");
//   printf("\nMatrix B:\n");
//   printMat(B,WB,HB);
//   printf("\n");
//   printf("\nMatrix C:\n");
//   printMat(C,WC,HC);
//   printf("\n");
// }

// __host__ int freeMatrices(float *A, float *B, float *C, float *AA, float *BB, float *CC){
//   free( A );  free( B );  free ( C );
//   hipblasStatus_t status = cublasFree(AA);
//   if (status != HIPBLAS_STATUS_SUCCESS) {
//     fprintf (stderr, "!!!! memory free error (A)\n");
//     return EXIT_FAILURE;
//   }
//   status = cublasFree(BB);
//   if (status != HIPBLAS_STATUS_SUCCESS) {
//     fprintf (stderr, "!!!! memory free error (B)\n");
//     return EXIT_FAILURE;
//   }
//   status = cublasFree(CC);
//   if (status != HIPBLAS_STATUS_SUCCESS) {
//     fprintf (stderr, "!!!! memory free error (C)\n");
//     return EXIT_FAILURE;
//   }
//   return EXIT_SUCCESS;
// }

int  main (int argc, char** argv) {
  hipblasStatus_t status;
  //cublasInit(); // Removed for transisting to cublas v2 API.

  double *A64 = initializeGroundtruthMat<double>(HA, WA, true, -1);
  double *B64 = initializeGroundtruthMat<double>(HB, WB, true, -1);

  // Initialize matrices A and B (2d arrays) based on the HA/WA and HB/WB to be filled with random data
  float *floatA = initializeFloatFromDouble(HA, WA, A64);
  float *floatB = initializeFloatFromDouble(HB, WB, B64);
  // Create arrays of C64 and it should contain the value of A64*B64, as groundtruth.
  double *C64 = initializeGroundtruthMat<double>(HC, WC, true, -1);

  if( A64 == 0 || B64 == 0 || C64 == 0){
    return EXIT_FAILURE;
  } else {
    // Since Ampere support IEEE-compliant FP64 computations, we use cublas for the computation.
    float *floatC = initializeGroundtruthMat<float>(HC, WC, false, 0);
    float *floatCFromCpu = initializeGroundtruthMat<float>(HC, WC, false, -1);
    float alpha = 1.0f;
    float beta = 0.0f;

    // TODO use initializeDeviceMemoryFromHostMemory to create AA from matrix A
    float *devA = initializeDeviceFloatFromHostFloat(HA, WA, floatA);
    // TODO use initializeDeviceMemoryFromHostMemory to create BB from matrix B
    float *devB = initializeDeviceFloatFromHostFloat(HB, WB, floatB);
    // TODO use initializeDeviceMemoryFromHostMemory to create CC from matrix C
    float *devC = initializeDeviceFloatFromHostFloat(HC, WC, floatC);

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // TODO perform Single-Precision Matrix to Matrix Multiplication, GEMM, on AA and BB and place results in CC
    status = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, HA, WB, WA, &alpha,
          devA, HA,
          devB, HB,
          &beta,
          devC, HC);
    if (status != HIPBLAS_STATUS_SUCCESS) {
      fprintf (stderr, "!!!! gemm error (A)\n");
      return EXIT_FAILURE;
    }

    floatC = retrieveDeviceMemory(HC, WC, devC, floatC);

    printf("==== A ====\n");
    printMat(floatA, WA, HA);
    printf("==== B ====\n");
    printMat(floatB, WB, HB);
    printf("==== C ====\n");
    printMat(floatC, WC, HC);
    //printMatrices(floatA, floatB, floatC);

    free(A64);
    free(B64);
    free(C64);
    free(floatA);
    free(floatB);
    free(floatC);

    hipFree(devA);
    hipFree(devB);
    hipFree(devC);
    
    /* Shutdown */
    status = hipblasDestroy(handle);
    if (status != HIPBLAS_STATUS_SUCCESS) {
      fprintf (stderr, "!!!! shutdown error (A)\n");
      return EXIT_FAILURE;
    }

    return EXIT_SUCCESS;
  }

}

