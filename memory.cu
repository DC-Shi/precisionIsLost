#include <stdlib.h>
#include <stdio.h>
#include <hipblas.h>
#include "memory.cuh"


__host__ float* initializeFloatFromDouble(int height, int width, double* inputDouble)
{
  float *ret = (float*)malloc(height * width * sizeof(float));

  for (int i = 0; i < height*width; i++)  ret[i] = inputDouble[i];

  return ret;
}



__host__ void printMatrices(float *A, float *B, float *C){
  printf("\nMatrix A:\n");
  printMat(A,WA,HA);
  printf("\n");
  printf("\nMatrix B:\n");
  printMat(B,WB,HB);
  printf("\n");
  printf("\nMatrix C:\n");
  printMat(C,WC,HC);
  printf("\n");
}


__host__ int freeMatrices(float *A, float *B, float *C, float *AA, float *BB, float *CC){
  free( A );  free( B );  free ( C );
  hipblasStatus_t status = cublasFree(AA);
  if (status != HIPBLAS_STATUS_SUCCESS) {
    fprintf (stderr, "!!!! memory free error (A)\n");
    return EXIT_FAILURE;
  }
  status = cublasFree(BB);
  if (status != HIPBLAS_STATUS_SUCCESS) {
    fprintf (stderr, "!!!! memory free error (B)\n");
    return EXIT_FAILURE;
  }
  status = cublasFree(CC);
  if (status != HIPBLAS_STATUS_SUCCESS) {
    fprintf (stderr, "!!!! memory free error (C)\n");
    return EXIT_FAILURE;
  }
  return EXIT_SUCCESS;
}


const char* cublasGetErrorString(hipblasStatus_t status)
{
    switch(status)
    {
        case HIPBLAS_STATUS_SUCCESS: return "HIPBLAS_STATUS_SUCCESS";
        case HIPBLAS_STATUS_NOT_INITIALIZED: return "HIPBLAS_STATUS_NOT_INITIALIZED";
        case HIPBLAS_STATUS_ALLOC_FAILED: return "HIPBLAS_STATUS_ALLOC_FAILED";
        case HIPBLAS_STATUS_INVALID_VALUE: return "HIPBLAS_STATUS_INVALID_VALUE"; 
        case HIPBLAS_STATUS_ARCH_MISMATCH: return "HIPBLAS_STATUS_ARCH_MISMATCH"; 
        case HIPBLAS_STATUS_MAPPING_ERROR: return "HIPBLAS_STATUS_MAPPING_ERROR";
        case HIPBLAS_STATUS_EXECUTION_FAILED: return "HIPBLAS_STATUS_EXECUTION_FAILED"; 
        case HIPBLAS_STATUS_INTERNAL_ERROR: return "HIPBLAS_STATUS_INTERNAL_ERROR"; 
    }
    return "unknown error";
}
